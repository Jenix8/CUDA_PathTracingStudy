#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <>

# include <GLFW/glfw3.h>
# include <glm/vec3.hpp>
# include <glm/vec4.hpp>
# include <glm/mat4x4.hpp>

# include <stdlib.h>
# include <stdio.h>
#include <vector>

#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ glm::vec3 color(const ray& r, hitable** world, hiprandState* local_rand_state) {
    ray cur_ray = r;
    glm::vec3 cur_attenuation = glm::vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            glm::vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return glm::vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            glm::vec3 unit_direction = glm::normalize(cur_ray.direction());
            float t = 0.5f * (unit_direction.y + 1.0f);
            glm::vec3 c = (1.0f - t) * glm::vec3(1.0, 1.0, 1.0) + t * glm::vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return glm::vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1000, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;

    hiprand_init(2000 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(glm::vec3* fb, int max_x, int max_y, int ns, camera** cam, hitable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];

    glm::vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }

    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(glm::vec3(0, -1000.0, -1), 1000, new lambertian(glm::vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                glm::vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2, new lambertian(glm::vec3(RND * RND, RND * RND, RND * RND)));
                    // d_list[i++] = new sphere(center, 0.2, new lambertian(glm::vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND))));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2, new metal(glm::vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
                    // d_list[i++] = new sphere(center, 0.2, new metal(glm::vec3(RND * RND, RND * RND, RND * RND), 0.5f * RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(glm::vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_list[i++] = new sphere(glm::vec3(-4, 1, 0), 1.0, new lambertian(glm::vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(glm::vec3(4, 1, 0), 1.0, new metal(glm::vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world = new hitable_list(d_list, 22 * 22 + 1 + 3);

        glm::vec3 lookfrom(13, 2, 3);
        glm::vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0;
        // (lookfrom - lookat).length();
        glm::length(lookfrom - lookat);
        float aperture = 0.1;
        *d_camera = new camera(lookfrom, lookat, glm::vec3(0, 1, 0), 30.0, float(nx) / float(ny), aperture, dist_to_focus);
    }
}

__global__ void free_world(hitable** d_list, hitable** d_world, camera** d_camera) {
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete ((sphere*)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}

void draw(glm::vec3* fb, int nx, int ny)
{
    static glm::vec3* pixels = new glm::vec3[nx * ny];

#pragma omp parallel for
    for (int x = 0; x < nx; x++)
    {
        for (int y = 0; y < ny; y++)
        {
            int index = x + y * nx;

            glm::vec3 color = fb[index];

            // gamma-correction r = 1/2
            pixels[index] = sqrt((color));
        }
    }

    glDrawPixels(nx, ny, GL_RGB, GL_FLOAT, pixels);
    glFlush();
}

void ExcuteCuda()
{
    int nx = 1200;
    int ny = 800;
    int ns = 20;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(glm::vec3);

    // allocate FB
    glm::vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> > (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    hitable** d_list;
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(hitable*)));
    hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    render_init << <blocks, threads >> > (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render << <blocks, threads >> > (fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    GLFWwindow* window;

    if (!glfwInit())
    {
        printf("Error: Initialize GLFW failure\n");;
        exit(EXIT_FAILURE);
    }


    window = glfwCreateWindow(nx, ny, "CPT", NULL, NULL);
    glfwGetFramebufferSize(window, &nx, &ny);

    if (!window) {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }

    glfwMakeContextCurrent(window);
    //srand(3211);

    // init();
    draw(fb, nx, ny);
    glfwSwapBuffers(window);
    draw(fb, nx ,ny);

    while (!glfwWindowShouldClose(window))
    {
        glfwPollEvents();
        glfwSwapBuffers(window);
    }

    glfwDestroyWindow(window);

    glfwTerminate();
    exit(EXIT_SUCCESS);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}